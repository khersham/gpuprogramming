
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *d_a, int *d_b, int *d_c){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  d_c[index] = d_a[index] + d_b[index];
}

int main(int argc, char ** argv){
  int N = 12;
  int size = N * sizeof(int);
  int a[N], b[N], c[N];
  int *d_a, *d_b, *d_c;


  //Alloc space for device
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  for (int i = 0; i< N; i++){
    a[i] = i;
    b[i] = 2*i;
  }

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<N,1>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  for(int i = 0; i < N; i++){
    printf("%i", c[i]);
    printf("\n");
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}